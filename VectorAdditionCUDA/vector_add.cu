// LeetGPU - Vector Addition

#include <hip/hip_runtime.h>
#include <iostream>

#define ROWS 2
#define COLS 5

__global__ void matrix_add(const float *A, const float *B, float *C, int rows, int cols)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < rows && col < cols)
    {
        int index = row * cols + col;
        C[index] = A[index] + B[index];
    }
}

void solve(const float *A, const float *B, float *C, int rows, int cols)
{
    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((cols + threadsPerBlock.x - 1) / threadsPerBlock.x,
                       (rows + threadsPerBlock.y - 1) / threadsPerBlock.y);

    matrix_add<<<blocksPerGrid, threadsPerBlock>>>(A, B, C, rows, cols);
    hipDeviceSynchronize();
}

int main()
{
    const int N = ROWS * COLS;
    float h_A[ROWS][COLS] = {{1, 2, 3, 4, 5},
                             {6, 7, 8, 9, 10}};

    float h_B[ROWS][COLS] = {{2, 4, 6, 8, 10},
                             {12, 14, 16, 18, 20}};

    float h_C[ROWS][COLS];

    float *d_A, *d_B, *d_C;
    hipMalloc((void **)&d_A, N * sizeof(float));
    hipMalloc((void **)&d_B, N * sizeof(float));
    hipMalloc((void **)&d_C, N * sizeof(float));

    hipMemcpy(d_A, h_A, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, N * sizeof(float), hipMemcpyHostToDevice);

    solve(d_A, d_B, d_C, ROWS, COLS);

    hipMemcpy(h_C, d_C, N * sizeof(float), hipMemcpyDeviceToHost);

    std::cout << "Matrix C:" << std::endl;
    for (int i = 0; i < ROWS; i++)
    {
        for (int j = 0; j < COLS; j++)
        {
            std::cout << h_C[i][j] << " ";
        }
        std::cout << std::endl;
    }

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return 0;
}
